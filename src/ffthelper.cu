#include "hip/hip_runtime.h"
#include "cuda_helper.h"
#include "ffthelper.h"
#include "utils.h"

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <math.h>
#include "hip/hip_complex.h"

#include <complex>
#include <vector>

using ComplexVec = std::vector<std::complex<float>>;

namespace refft {

// Modular multiplication a * N mod p
// In: a[np][N]

// TRANSPOSE CONSTANT
const int T_SMEM_SIZE_1 = 16;
const int T_BLOCK_ROW_1 = 16;
const int T_SMEM_SIZE_2 = 16;
const int T_BLOCK_ROW_2 = 16;
const int T_nx = 256;
const int T_ny = 128;

// FFT CONSTANT
const int FFT1_SIZE = 128;
const int FFT2_SIZE = 256;
const int BLOCK_DIM_1 = 32;
const int GRID_DIM_1 = 256;
const int BLOCK_DIM_2 = 64;
const int GRID_DIM_2 = 128;

__device__ hipFloatComplex twiddle(const float expr) {
  hipFloatComplex res;
  sincosf(expr, &res.y, &res.x);
  return res;
}

__device__ void twd(hipFloatComplex *x, hipFloatComplex *data, const int N) {
  const double theta = -2 * M_PI/N;
  hipFloatComplex w = twiddle(theta);
  
  //hipFloatComplex w1 = data[threadIdx.x * blockIdx.x]; //twiddle(theta * threadIdx.x * blockIdx.x);
  //hipFloatComplex w2 = data[(threadIdx.x + BLOCK_DIM_1) * blockIdx.x]; //twiddle(theta * (threadIdx.x + BLOCK_DIM_1) * blockIdx.x);
  //hipFloatComplex w3 = data[(threadIdx.x + 2 * BLOCK_DIM_1) * blockIdx.x]; //twiddle(theta * (threadIdx.x + 2 * BLOCK_DIM_1) * blockIdx.x);
  //hipFloatComplex w4 = data[(threadIdx.x + 3 * BLOCK_DIM_1) * blockIdx.x]; //twiddle(theta * (threadIdx.x + 3 * BLOCK_DIM_1) * blockIdx.x); 
 
  hipFloatComplex w1 = twiddle(theta * threadIdx.x * blockIdx.x);
  hipFloatComplex w2 = twiddle(theta * (threadIdx.x + BLOCK_DIM_1) * blockIdx.x);
  hipFloatComplex w3 = twiddle(theta * (threadIdx.x + 2 * BLOCK_DIM_1) * blockIdx.x);
  hipFloatComplex w4 = twiddle(theta * (threadIdx.x + 3 * BLOCK_DIM_1) * blockIdx.x); 

  hipFloatComplex a = hipCmulf(w1, x[threadIdx.x]);
  hipFloatComplex b = hipCmulf(w2, x[threadIdx.x + BLOCK_DIM_1]);
  hipFloatComplex c = hipCmulf(w3, x[threadIdx.x + 2 * BLOCK_DIM_1]);
  hipFloatComplex d = hipCmulf(w4, x[threadIdx.x + 3 * BLOCK_DIM_1]);
  __syncthreads();

  x[threadIdx.x] = a;
  x[threadIdx.x + BLOCK_DIM_1] = b;
  x[threadIdx.x + 2 * BLOCK_DIM_1] = c;
  x[threadIdx.x + 3 * BLOCK_DIM_1] = d;
  __syncthreads();
}

// RADIX-4 STOCKHAM ALGORITHM FFT
__device__ void fft_radix4_even(int n, hipFloatComplex *x, hipFloatComplex *t) {
  hipFloatComplex j = make_hipFloatComplex(0, 1);// j.x = 0; j.y = 1;
  int s = 1;
  int m = n;
  int k = 0;
  const int n1 = n/4;
  const int n2 = n/2;
  const int n3 = n1 + n2;

  for(int i = 0; i < 4; i++) {
    double theta = 2 * M_PI / m;
    int q = threadIdx.x % s;
    int p = (threadIdx.x - q) >> k;

    hipFloatComplex w1 = twiddle(- p * theta);
    hipFloatComplex w2 = hipCmulf(w1, w1);
    hipFloatComplex w3 = hipCmulf(w1, w2);
    //hipFloatComplex w1 = t[256 * p / m];
    //hipFloatComplex w2 = hipCmulf(w1, w1);
    //hipFloatComplex w3 = hipCmulf(w1, w2);
   
    hipFloatComplex a = x[q + s * p];
    hipFloatComplex b = x[q + s * p + n1];
    hipFloatComplex c = x[q + s * p + n2];
    hipFloatComplex d = x[q + s * p + n3];
    hipFloatComplex temp1_even = hipCaddf(a, c);
    hipFloatComplex temp1_odd = hipCsubf(a, c);
    hipFloatComplex temp2_even = hipCaddf(b, d);
    hipFloatComplex temp2_odd = hipCmulf(j, hipCsubf(b, d));
    __syncthreads();
    
    //add
    x[q + s * 4 * p] = hipCaddf(temp1_even, temp2_even);
    x[q + s * (4 * p + 1)] = hipCmulf(w1, hipCsubf(temp1_odd, temp2_odd));
    x[q + s * (4 * p + 2)] = hipCmulf(w2, hipCsubf(temp1_even, temp2_even));
    x[q + s * (4 * p + 3)] = hipCmulf(w3, hipCaddf(temp1_odd, temp2_odd));
    __syncthreads();

    s = s << 2;
    m = m >> 2;
    k = k + 2;
  }
}

// RADIX-4 STOCKHAM ALGORITHM FFT
__device__ void fft_radix4_odd(int n, hipFloatComplex *x, hipFloatComplex *t) {
  hipFloatComplex j = make_hipFloatComplex(0, 1);
  int s = 1;
  int m = n;
  int k = 0;
  const int n1 = n/4;
  const int n2 = n/2;
  const int n3 = n1 + n2;

  for(int i = 0; i < 3; i++) {
    double theta = 2*M_PI/m;
    int q = threadIdx.x % s;
    int p = (threadIdx.x - q) >> k;
    
    hipFloatComplex w1 = twiddle(- p * theta);
    hipFloatComplex w2 = hipCmulf(w1, w1);
    hipFloatComplex w3 = hipCmulf(w1, w2);
    //hipFloatComplex w1 = t[256 * p / m];
    //hipFloatComplex w2 = hipCmulf(w1, w1);
    //hipFloatComplex w3 = hipCmulf(w1, w2);
    
    hipFloatComplex a = x[q + s * p];
    hipFloatComplex b = x[q + s * p + n1];
    hipFloatComplex c = x[q + s * p + n2];
    hipFloatComplex d = x[q + s * p + n3];
    hipFloatComplex temp1_even = hipCaddf(a, c);
    hipFloatComplex temp1_odd = hipCsubf(a, c);
    hipFloatComplex temp2_even = hipCaddf(b, d);
    hipFloatComplex temp2_odd = hipCmulf(j, hipCsubf(b, d));
    __syncthreads();

    x[q + s * 4 * p] = hipCaddf(temp1_even, temp2_even);
    x[q + s * (4 * p + 1)] = hipCmulf(w1, hipCsubf(temp1_odd, temp2_odd));
    x[q + s * (4 * p + 2)] = hipCmulf(w2, hipCsubf(temp1_even, temp2_even));
    x[q + s * (4 * p + 3)] = hipCmulf(w3, hipCaddf(temp1_odd, temp2_odd));
    __syncthreads();    

    s = s << 2;
    m = m >> 2;
    k = k + 2;
  }

  hipFloatComplex a = x[threadIdx.x];
  hipFloatComplex b = x[threadIdx.x + (n >> 1)];
  hipFloatComplex c = x[threadIdx.x + (n >> 2)];
  hipFloatComplex d = x[threadIdx.x + (n >> 1) + (n >> 2)];
  hipFloatComplex temp1_even = hipCaddf(a, b);
  hipFloatComplex temp1_odd = hipCsubf(a, b);
  hipFloatComplex temp2_even = hipCaddf(c, d);
  hipFloatComplex temp2_odd = hipCsubf(c, d);
  __syncthreads();
  
  // twiddle
  //const double theta = -2 * M_PI/N;
  //hipFloatComplex w1 = twiddle(theta * threadIdx.x * blockIdx.x);
  //hipFloatComplex w2 = twiddle(theta * (threadIdx.x + BLOCK_DIM_1) * blockIdx.x);
  //hipFloatComplex w3 = twiddle(theta * (threadIdx.x + 2 * BLOCK_DIM_1) * blockIdx.x);
  //hipFloatComplex w4 = twiddle(theta * (threadIdx.x + 3 * BLOCK_DIM_1) * blockIdx.x); 
  
  //temp1_even = hipCmulf(w1, temp1_even);
  //temp2_even = hipCmulf(w2, temp2_even);
  //temp1_odd = hipCmulf(w3, temp1_odd);
  //temp2_odd = hipCmulf(w4, temp2_odd);
  //__syncthreads();

  x[threadIdx.x] =  temp1_even;
  x[threadIdx.x + (n >> 1)] = temp1_odd;
  x[threadIdx.x + (n >> 2)] = temp2_even;
  x[threadIdx.x + (n >> 1) + (n >> 2)] = temp2_odd;

  __syncthreads();
}

__device__ void butt_fft(hipFloatComplex *a, hipFloatComplex *b,
                         hipFloatComplex w) {
  hipFloatComplex U = hipCmulf(*b, w);
  *b = hipCsubf(*a, U);
  *a = hipCaddf(*a, U);
}

__global__ void Twiddle_Factor(hipFloatComplex *twiddle_factor, const int N) {
  const float theta = - M_PI * (threadIdx.x + blockIdx.x * blockDim.x) / N;
  const hipFloatComplex c = twiddle(theta);
  twiddle_factor[threadIdx.x + blockIdx.x * blockDim.x] = c;
}

__global__ void Transpose1(hipFloatComplex *a)
{
  __shared__ hipFloatComplex smem[T_SMEM_SIZE_1][T_SMEM_SIZE_1 + 1];
    
  int x = blockIdx.x * T_SMEM_SIZE_1 + threadIdx.x;
  int y = blockIdx.y * T_SMEM_SIZE_1 + threadIdx.y;
  int width_x = gridDim.x * T_SMEM_SIZE_1;
  int width_y = gridDim.y * T_SMEM_SIZE_1;
  
  //for (int i = 0; i < T_SMEM_SIZE_1; i += T_BLOCK_ROW_1)
  //  smem[threadIdx.y + i][threadIdx.x] = a[(y + i) * width_x + x];
  smem[threadIdx.y][threadIdx.x] = a[y * width_x + x];
  __syncthreads();

  x = blockIdx.y * T_SMEM_SIZE_1 + threadIdx.x;
  y = blockIdx.x * T_SMEM_SIZE_1 + threadIdx.y;

  //for (int i = 0; i < T_SMEM_SIZE_1; i += T_BLOCK_ROW_1)
  a[y * width_y + x] = smem[threadIdx.x][threadIdx.y];
}

__global__ void Transpose2(hipFloatComplex *a)
{
  __shared__ hipFloatComplex smem[T_SMEM_SIZE_2][T_SMEM_SIZE_2 + 1];

  int x = blockIdx.x * T_SMEM_SIZE_2 + threadIdx.x;
  int y = blockIdx.y * T_SMEM_SIZE_2 + threadIdx.y;
  int width_x = gridDim.x * T_SMEM_SIZE_2;
  int width_y = gridDim.y * T_SMEM_SIZE_2;

  for (int i = 0; i < T_SMEM_SIZE_2; i += T_BLOCK_ROW_2)
    smem[threadIdx.y + i][threadIdx.x] = a[(y + i) * width_x + x];
  //smem[threadIdx.y][threadIdx.x] = a[y * width_x + x];
  __syncthreads();
 
  x = blockIdx.y * T_SMEM_SIZE_2 + threadIdx.x;
  y = blockIdx.x * T_SMEM_SIZE_2 + threadIdx.y;

  for (int i = 0; i < T_SMEM_SIZE_2; i += T_BLOCK_ROW_2)
    a[(y + i) * width_y + x] = smem[threadIdx.x][threadIdx.y + i];
  //a[y * width_y + x] = smem[threadIdx.x][threadIdx.y];
}

__global__ void Fft(hipFloatComplex *a, const int m, const int N) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (N / 2);
       i += blockDim.x * gridDim.x) {
    // index in N/2 range
    int N_idx = i % (N / 2);
    // i'th block
    int m_idx = N_idx / m;
    // base address
    hipFloatComplex *a_np = a;
    int t_idx = N_idx % m;
    hipFloatComplex *a_x = a_np + 2 * m_idx * m + t_idx;
    hipFloatComplex *a_y = a_x + m;
    hipFloatComplex w = twiddle(-M_PI * (double)t_idx / (double)m);
    butt_fft(a_x, a_y, w);
  }
}

__global__ void FftWithTwiddle_Radix4(hipFloatComplex *a, hipFloatComplex *t, hipFloatComplex *data, const int N) {
      // shared memory
      extern __shared__ hipFloatComplex x[];
      __shared__ hipFloatComplex tf[64];
      // global memory -> shared memory without shared memory bank conflict
      x[threadIdx.x]                   = a[blockIdx.x * FFT1_SIZE + threadIdx.x];
      x[threadIdx.x + BLOCK_DIM_1]     = a[blockIdx.x * FFT1_SIZE + BLOCK_DIM_1 + threadIdx.x];
      x[threadIdx.x + 2 * BLOCK_DIM_1] = a[blockIdx.x * FFT1_SIZE + 2 * BLOCK_DIM_1 + threadIdx.x];
      x[threadIdx.x + 3 * BLOCK_DIM_1] = a[blockIdx.x * FFT1_SIZE + 3 * BLOCK_DIM_1 + threadIdx.x];

      tf[threadIdx.x]                 = t[threadIdx.x];
      tf[threadIdx.x + blockDim.x]    = t[threadIdx.x + blockDim.x];
      __syncthreads();

      // FFT + Twiddle
      fft_radix4_odd(FFT1_SIZE, x, tf);
      twd(x, data, N);

      // shared memory -> global memory without shared memory bank conflict
      a[blockIdx.x * FFT1_SIZE + threadIdx.x]                   = x[threadIdx.x];
      a[blockIdx.x * FFT1_SIZE + BLOCK_DIM_1 + threadIdx.x]     = x[threadIdx.x + BLOCK_DIM_1];
      a[blockIdx.x * FFT1_SIZE + 2 * BLOCK_DIM_1 + threadIdx.x] = x[threadIdx.x + 2 * BLOCK_DIM_1];
      a[blockIdx.x * FFT1_SIZE + 3 * BLOCK_DIM_1 + threadIdx.x] = x[threadIdx.x + 3 * BLOCK_DIM_1];
}

__global__ void FftWithoutTwiddle_Radix4(hipFloatComplex *a, hipFloatComplex *t) {
  //for(int i = 0; i < 2; i++) {  
    // shared memory
    extern __shared__ hipFloatComplex x[];
    __shared__ hipFloatComplex tf[64];

    // global memory -> shared memory without shared memory bank conflict
    x[threadIdx.x]                   = a[blockIdx.x * FFT2_SIZE + threadIdx.x];
    x[threadIdx.x + BLOCK_DIM_2]     = a[blockIdx.x * FFT2_SIZE + BLOCK_DIM_2 + threadIdx.x];
    x[threadIdx.x + 2 * BLOCK_DIM_2] = a[blockIdx.x * FFT2_SIZE + 2 * BLOCK_DIM_2 + threadIdx.x];
    x[threadIdx.x + 3 * BLOCK_DIM_2] = a[blockIdx.x * FFT2_SIZE + 3 * BLOCK_DIM_2 + threadIdx.x];
    
    tf[threadIdx.x]                 = t[threadIdx.x];
    __syncthreads();

    // FFT
    fft_radix4_even(FFT2_SIZE, x, tf);

    // shared memory -> global memory without shared memory bank conflict
    a[blockIdx.x * FFT2_SIZE + threadIdx.x]                   = x[threadIdx.x];
    a[blockIdx.x * FFT2_SIZE + BLOCK_DIM_2 + threadIdx.x]     = x[threadIdx.x + BLOCK_DIM_2];
    a[blockIdx.x * FFT2_SIZE + 2 * BLOCK_DIM_2 + threadIdx.x] = x[threadIdx.x + 2 * BLOCK_DIM_2];
    a[blockIdx.x * FFT2_SIZE + 3 * BLOCK_DIM_2 + threadIdx.x] = x[threadIdx.x + 3 * BLOCK_DIM_2];
  //}
}

__global__ void FftStudent(hipFloatComplex *a, const int m, const int N) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (N / 2) * 1;
       i += blockDim.x * gridDim.x) {
    // index in N/2 range
    int N_idx = i % (N / 2);
    // i'th block
    int m_idx = N_idx / m;
    // base address
    hipFloatComplex *a_np = a;
    int t_idx = N_idx % m;
    hipFloatComplex *a_x = a_np + 2 * m_idx * m + t_idx;
    hipFloatComplex *a_y = a_x + m;
    hipFloatComplex w = twiddle(-M_PI * (double)t_idx / (double)m);
    butt_fft(a_x, a_y, w);
  }
}

__device__ void butt_ifft(hipFloatComplex *a, hipFloatComplex *b,
                          hipFloatComplex w) {
  hipFloatComplex T = hipCsubf(*a, *b);
  *a = hipCaddf(*a, *b);
  (*a).x /= 2.0;
  (*a).y /= 2.0;
  *b = hipCmulf(T, w);
  (*b).x /= 2.0;
  (*b).y /= 2.0;
}

__global__ void Ifft(hipFloatComplex *a, const int m, const int N) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (N / 2);
       i += blockDim.x * gridDim.x) {
    // index in N/2 range
    int N_idx = i % (N / 2);
    // i'th block
    int m_idx = N_idx / m;
    // base address
    hipFloatComplex *a_np = a;
    int t_idx = N_idx % m;
    hipFloatComplex *a_x = a_np + 2 * m_idx * m + t_idx;
    hipFloatComplex *a_y = a_x + m;
    hipFloatComplex w = twiddle(M_PI * (double)t_idx / (double)m);
    butt_ifft(a_x, a_y, w);
  }
}

__global__ void IfftStudent(hipFloatComplex *a, const int m, const int N) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (N / 2);
       i += blockDim.x * gridDim.x) {
    // index in N/2 range
    int N_idx = i % (N / 2);
    // i'th block
    int m_idx = N_idx / m;
    // base address
    hipFloatComplex *a_np = a;
    int t_idx = N_idx % m;
    hipFloatComplex *a_x = a_np + 2 * m_idx * m + t_idx;
    hipFloatComplex *a_y = a_x + m;
    hipFloatComplex w = twiddle(M_PI * (double)t_idx / (double)m);
    butt_ifft(a_x, a_y, w);
  }
}

__global__ void bitReverse(std::complex<float> *a, int N) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (N);
       i += blockDim.x * gridDim.x) {
    int logN = __log2f(N);
    int N_idx = i % N;
    std::complex<float> *a_x = a;
    int revN = __brev(N_idx) >> (32 - logN);
    if (revN > N_idx) {
      std::complex<float> temp = a_x[N_idx];
      a_x[N_idx] = a_x[revN];
      a_x[revN] = temp;
    }
  }
}

__device__ hipFloatComplex Cmul(hipFloatComplex a, hipFloatComplex b) {
  float temp = double(a.x) * b.x - double(a.y) * b.y;
  float temp2 = double(a.x) * b.y + double(a.y) * b.x;
  hipFloatComplex res;
  res.x = temp;
  res.y = temp2;
  return res;
}

__global__ void Hadamard(hipFloatComplex *a, hipFloatComplex *b, int N) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (N);
       i += blockDim.x * gridDim.x) {
    int N_idx = i % N;
    hipFloatComplex *a_x = a;
    hipFloatComplex *b_x = b;
    a_x[N_idx] = Cmul(a_x[N_idx], b_x[N_idx]);
  }
}

void FftHelper::ExecFft(std::complex<float> *a, int N) {
  dim3 blockDim(refft::FFTblocksize);
  dim3 gridDim(N/2/refft::FFTblocksize);
  bitReverse<<<gridDim, blockDim>>>(a,N);
  for (int i = 1; i < N; i *= 2) {
    Fft<<<gridDim, blockDim>>>((hipFloatComplex *)a, i, N);
    CudaCheckError();
  }
  CudaCheckError();
}

/*
void FftHelper::ExecStudentFft(std::complex<float> *a, int N) {
  //dim3 blockDim(refft::FFTblocksize);
  //dim3 gridDim(N/2/refft::FFTblocksize);
  dim3 blockDim1(256);
  dim3 gridDim1(64);

  bitReverse<<<gridDim1, blockDim1>>>(a,N);
  for (int i = 1; i < N; i *= 2) {
    FftStudent<<<gridDim1, blockDim1>>>((hipFloatComplex *)a, i, N);
    CudaCheckError();
  }
  CudaCheckError();
}
*/

void FftHelper::ExecStudentFft(std::complex<float> *a, std::complex<float> *twiddle_factor, std::complex<float> *data, int N){
  dim3 gridDim1(T_nx/T_SMEM_SIZE_1, T_ny/T_SMEM_SIZE_1, 1);
  dim3 blockDim1(T_SMEM_SIZE_1, T_BLOCK_ROW_1, 1);
  dim3 blockDim2(BLOCK_DIM_1);
  dim3 gridDim2(GRID_DIM_1);
  dim3 gridDim3(T_ny/T_SMEM_SIZE_2, T_nx/T_SMEM_SIZE_2, 1);
  dim3 blockDim3(T_SMEM_SIZE_2, T_BLOCK_ROW_2, 1);
  dim3 blockDim4(BLOCK_DIM_2);
  dim3 gridDim4(GRID_DIM_2);
  dim3 gridDim5(T_nx/T_SMEM_SIZE_1, T_ny/T_SMEM_SIZE_1, 1);
  dim3 blockDim5(T_SMEM_SIZE_1, T_BLOCK_ROW_1, 1);
  
  //Twiddle_Factor<<<16, 4>>>((hipFloatComplex *)twiddle_factor, 128);
  //Twiddle_Factor<<<32, 1024>>>((hipFloatComplex *)data, N/2);
  Transpose1<<<gridDim1, blockDim1>>>((hipFloatComplex *)a);
  FftWithTwiddle_Radix4<<<gridDim2, blockDim2, BLOCK_DIM_1 * 4 * sizeof(hipFloatComplex)>>>((hipFloatComplex *)a, (hipFloatComplex *)twiddle_factor, (hipFloatComplex *)data, N);
  CudaCheckError();
  Transpose2<<<gridDim3, blockDim3>>>((hipFloatComplex *)a);
  FftWithoutTwiddle_Radix4<<<gridDim4, blockDim4, BLOCK_DIM_2 * 4 * sizeof(hipFloatComplex)>>>((hipFloatComplex *)a, (hipFloatComplex *)twiddle_factor);
  CudaCheckError();
  Transpose1<<<gridDim5, blockDim5>>>((hipFloatComplex *)a);
  CudaCheckError();
}

void FftHelper::ExecIfft(std::complex<float> *a, int N) {
  dim3 blockDim(refft::iFFTblocksize);
  dim3 gridDim(N/2/refft::iFFTblocksize);
  for (int i = N / 2; i > 0; i >>= 1) {
    Ifft<<<gridDim, blockDim>>>((hipFloatComplex *)a, i, N);
  }
  bitReverse<<<gridDim, blockDim>>>(a, N);
  CudaCheckError();
}

void FftHelper::ExecStudentIfft(std::complex<float> *a, int N) {
  dim3 blockDim(refft::iFFTblocksize);
  dim3 gridDim(N/2/refft::iFFTblocksize);
  for (int i = N / 2; i > 0; i >>= 1) {
    IfftStudent<<<gridDim, blockDim>>>((hipFloatComplex *)a, i, N);
  }
  bitReverse<<<gridDim, blockDim>>>(a, N);
  CudaCheckError();
}

void FftHelper::Mult(std::complex<float> *a, std::complex<float> *b, int N) {
  dim3 blockDim(refft::iFFTblocksize);
  dim3 gridDim(N/refft::iFFTblocksize);
  Hadamard<<<gridDim, blockDim>>>((hipFloatComplex*)a,(hipFloatComplex*)b, N);  
  CudaCheckError();
}
}  // namespace refft
