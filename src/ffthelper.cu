#include "hip/hip_runtime.h"
#include "cuda_helper.h"
#include "ffthelper.h"
#include "utils.h"

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <math.h>
#include "hip/hip_complex.h"

#include <complex>
#include <vector>

using ComplexVec = std::vector<std::complex<float>>;

namespace refft {

// Modular multiplication a * N mod p
// In: a[np][N]

// TWIDDLE CONSTANT
const int TWIDDLE_GRID_DIM = 256;
const int TWIDDLE_BLOCK_DIM = 128;

// TRANSPOSE CONSTANT
const int T_SMEM_SIZE = 8;
const int T_BLOCK_ROW = 4;
const int T_nx = 256;
const int T_ny = 128;

// FFT CONSTANT
const int FFT1_SIZE = 128;
const int FFT2_SIZE = 256;
const int BLOCK_DIM_1 = 32;
const int GRID_DIM_1 = 256;
const int BLOCK_DIM_2 = 64;
const int GRID_DIM_2 = 128;

__device__ hipFloatComplex twiddle(const float expr) {
  hipFloatComplex res;
  sincosf(expr, &res.y, &res.x);
  return res;
}

// RADIX-8 STOCKHAM ALGORITHM FFT
__device__ void fft_radix8(int n, hipFloatComplex *x, hipFloatComplex *constant) {
  const hipFloatComplex j = {0, 1};
  const hipFloatComplex c1 = {0.70711, 0.70711};
  const hipFloatComplex c2 = {0.70711, -0.70711};
  int s = 1;
  int m = n;

  const int n1 = n / 8;
  const int n2 = n / 4;
  const int n3 = n1 + n2;
  const int n4 = n / 2;
  const int n5 = n1 + n4;
  const int n6 = n3 + n3;
  const int n7 = n3 + n4;

  for(int i = 0; i < 3; i++) {
    int q = threadIdx.x % s;
    int p = (threadIdx.x - q) / s;

    hipFloatComplex w1 = constant[512 * p / m];
    hipFloatComplex w2 = hipCmulf(w1, w1);
    hipFloatComplex w3 = constant[512 * 3 * p / m];
    hipFloatComplex w4 = hipCmulf(w2, w2);
    hipFloatComplex w5 = constant[512 * 5 * p / m];
    hipFloatComplex w6 = hipCmulf(w3, w3);
    hipFloatComplex w7 = constant[517 * 7 * p / m];

    hipFloatComplex a = x[q + s * p];
    hipFloatComplex b = x[q + s * p + n1];
    hipFloatComplex c = x[q + s * p + n2];
    hipFloatComplex d = x[q + s * p + n3];
    hipFloatComplex e = x[q + s * p + n4];
    hipFloatComplex f = x[q + s * p + n5];
    hipFloatComplex g = x[q + s * p + n6];
    hipFloatComplex h = x[q + s * p + n7];

    hipFloatComplex ae0 = hipCaddf(a, e);
    hipFloatComplex ae1 = hipCsubf(a, e);
    hipFloatComplex cg0 = hipCaddf(c, g);
    hipFloatComplex cg1 = hipCmulf(j, hipCsubf(c, g));
    hipFloatComplex bf0 = hipCaddf(b, f);
    hipFloatComplex bf1 = hipCsubf(b, f);
    hipFloatComplex dh0 = hipCaddf(d, h);
    hipFloatComplex dh1 = hipCmulf(j, hipCsubf(d, h));

    hipFloatComplex aceg0 = hipCaddf(ae0, cg0);
    hipFloatComplex aceg1 = hipCsubf(ae1, cg1);
    hipFloatComplex aceg2 = hipCsubf(ae0, cg0);
    hipFloatComplex aceg3 = hipCaddf(ae1, cg1);
    hipFloatComplex bdfh0 = hipCaddf(bf0, dh0);
    hipFloatComplex bdfh1 = hipCmulf(c1, hipCsubf(bf1, dh1));
    hipFloatComplex bdfh2 = hipCmulf(j, hipCsubf(bf0, dh0));
    hipFloatComplex bdfh3 = hipCmulf(c2, hipCaddf(bf1, dh1));
    __syncthreads();

    x[q + s * 8 * p] = hipCaddf(aceg0, bdfh0);
    x[q + s * (8 * p + 1)] = hipCmulf(w1, hipCsubf(aceg1, bdfh1));
    x[q + s * (8 * p + 2)] = hipCmulf(w2, hipCsubf(aceg2, bdfh2));
    x[q + s * (8 * p + 3)] = hipCmulf(w3, hipCsubf(aceg3, bdfh3));
    x[q + s * (8 * p + 4)] = hipCmulf(w4, hipCsubf(aceg0, bdfh0));
    x[q + s * (8 * p + 5)] = hipCmulf(w5, hipCaddf(aceg1, bdfh1));
    x[q + s * (8 * p + 6)] = hipCmulf(w6, hipCaddf(aceg2, bdfh2));
    x[q + s * (8 * p + 7)] = hipCmulf(w7, hipCaddf(aceg3, bdfh3));
    __syncthreads();

    s = s << 3;
    m = m >> 3;
  }
}

// RADIX-4 STOCKHAM ALGORITHM FFT
__device__ void fft_radix4_even(int n, hipFloatComplex *x, hipFloatComplex *constant) {
  const hipFloatComplex j = {0, 1};
  int s = 1;
  int m = n;
  int k = 0;
  const int n1 = n/4;
  const int n2 = n/2;
  const int n3 = n1 + n2;

  for(int i = 0; i < 4; i++) {
    int q = threadIdx.x % s;
    int p = (threadIdx.x - q) >> k;

    hipFloatComplex w1 = constant[256 * p / m];
    hipFloatComplex w2 = hipCmulf(w1, w1);
    hipFloatComplex w3 = hipCmulf(w1, w2);
    __syncthreads();

    hipFloatComplex a = x[q + s * p];
    hipFloatComplex b = x[q + s * p + n1];
    hipFloatComplex c = x[q + s * p + n2];
    hipFloatComplex d = x[q + s * p + n3];
    hipFloatComplex temp1_even = hipCaddf(a, c);
    hipFloatComplex temp1_odd = hipCsubf(a, c);
    hipFloatComplex temp2_even = hipCaddf(b, d);
    hipFloatComplex temp2_odd = hipCmulf(j, hipCsubf(b, d));
    __syncthreads();
    
    x[q + s * 4 * p] = hipCaddf(temp1_even, temp2_even);
    x[q + s * (4 * p + 1)] = hipCmulf(w1, hipCsubf(temp1_odd, temp2_odd));
    x[q + s * (4 * p + 2)] = hipCmulf(w2, hipCsubf(temp1_even, temp2_even));
    x[q + s * (4 * p + 3)] = hipCmulf(w3, hipCaddf(temp1_odd, temp2_odd));
    __syncthreads();

    s = s << 2;
    m = m >> 2;
    k = k + 2;
  }
}

// RADIX-4 STOCKHAM ALGORITHM FFT
__device__ void fft_radix4_odd(int n, hipFloatComplex *x, hipFloatComplex *constant, hipFloatComplex *t) {
  const hipFloatComplex j = {0, 1};
  int s = 1;
  int m = n;
  int k = 0;
  const int n1 = n/4;
  const int n2 = n/2;
  const int n3 = n1 + n2;

  for(int i = 0; i < 3; i++) {
    int q = threadIdx.x % s;
    int p = (threadIdx.x - q) >> k;
    
    hipFloatComplex w1 = constant[256 * p / m];
    hipFloatComplex w2 = hipCmulf(w1, w1);
    hipFloatComplex w3 = hipCmulf(w1, w2);
    
    hipFloatComplex a = x[q + s * p];
    hipFloatComplex b = x[q + s * p + n1];
    hipFloatComplex c = x[q + s * p + n2];
    hipFloatComplex d = x[q + s * p + n3];
    hipFloatComplex temp1_even = hipCaddf(a, c);
    hipFloatComplex temp1_odd = hipCsubf(a, c);
    hipFloatComplex temp2_even = hipCaddf(b, d);
    hipFloatComplex temp2_odd = hipCmulf(j, hipCsubf(b, d));
    __syncthreads();

    x[q + s * 4 * p] = hipCaddf(temp1_even, temp2_even);
    x[q + s * (4 * p + 1)] = hipCmulf(w1, hipCsubf(temp1_odd, temp2_odd));
    x[q + s * (4 * p + 2)] = hipCmulf(w2, hipCsubf(temp1_even, temp2_even));
    x[q + s * (4 * p + 3)] = hipCmulf(w3, hipCaddf(temp1_odd, temp2_odd));
    __syncthreads();    

    s = s << 2;
    m = m >> 2;
    k = k + 2;
  }

  hipFloatComplex a = x[threadIdx.x];
  hipFloatComplex b = x[threadIdx.x + n2];
  hipFloatComplex c = x[threadIdx.x + n1];
  hipFloatComplex d = x[threadIdx.x + n3];
  hipFloatComplex temp1_even = hipCaddf(a, b);
  hipFloatComplex temp1_odd = hipCsubf(a, b);
  hipFloatComplex temp2_even = hipCaddf(c, d);
  hipFloatComplex temp2_odd = hipCsubf(c, d);
  __syncthreads();
  
  // twiddle
  hipFloatComplex w1 = t[threadIdx.x * blockIdx.x];
  hipFloatComplex w = t[BLOCK_DIM_1 * blockIdx.x];
  __syncthreads();

  hipFloatComplex w2 = hipCmulf(w1, w);//twiddle(theta * (threadIdx.x + BLOCK_DIM_1) * blockIdx.x);
  hipFloatComplex w3 = hipCmulf(w2, w);//twiddle(theta * (threadIdx.x + 2 * BLOCK_DIM_1) * blockIdx.x);
  hipFloatComplex w4 = hipCmulf(w, hipCmulf(w, w2));//twiddle(theta * (threadIdx.x + 3 * BLOCK_DIM_1) * blockIdx.x); 
  __syncthreads();  

  temp1_even = hipCmulf(w1, temp1_even);
  temp2_even = hipCmulf(w2, temp2_even);
  temp1_odd = hipCmulf(w3, temp1_odd);
  temp2_odd = hipCmulf(w4, temp2_odd);
  __syncthreads();

  x[threadIdx.x] =  temp1_even;
  x[threadIdx.x + n2] = temp1_odd;
  x[threadIdx.x + n1] = temp2_even;
  x[threadIdx.x + n3] = temp2_odd;
  __syncthreads();
}

__device__ void butt_fft(hipFloatComplex *a, hipFloatComplex *b,
                         hipFloatComplex w) {
  hipFloatComplex U = hipCmulf(*b, w);
  *b = hipCsubf(*a, U);
  *a = hipCaddf(*a, U);
}

__global__ void Cal(hipFloatComplex *a, const int N) {
  const float theta = - M_PI * (threadIdx.x + blockIdx.x * blockDim.x) / N;
  const hipFloatComplex c = twiddle(theta);
  a[threadIdx.x + blockIdx.x * blockDim.x] = c;
}

__global__ void Transpose(hipFloatComplex *a)
{
  __shared__ hipFloatComplex smem[T_SMEM_SIZE][T_SMEM_SIZE + 1];
    
  int x = blockIdx.x * T_SMEM_SIZE + threadIdx.x;
  int y = blockIdx.y * T_SMEM_SIZE + threadIdx.y;
  int width_x = gridDim.x * T_SMEM_SIZE;
  int width_y = gridDim.y * T_SMEM_SIZE;
  
  for (int i = 0; i < T_SMEM_SIZE; i += T_BLOCK_ROW)
    smem[threadIdx.y + i][threadIdx.x] = a[(y + i) * width_x + x];
  __syncthreads();

  x = blockIdx.y * T_SMEM_SIZE + threadIdx.x;
  y = blockIdx.x * T_SMEM_SIZE + threadIdx.y;

  for (int i = 0; i < T_SMEM_SIZE; i += T_BLOCK_ROW)
    a[(y + i) * width_y + x] = smem[threadIdx.x][threadIdx.y + i];
}

__global__ void Fft(hipFloatComplex *a, const int m, const int N) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (N / 2);
       i += blockDim.x * gridDim.x) {
    // index in N/2 range
    int N_idx = i % (N / 2);
    // i'th block
    int m_idx = N_idx / m;
    // base address
    hipFloatComplex *a_np = a;
    int t_idx = N_idx % m;
    hipFloatComplex *a_x = a_np + 2 * m_idx * m + t_idx;
    hipFloatComplex *a_y = a_x + m;
    hipFloatComplex w = twiddle(-M_PI * (double)t_idx / (double)m);
    butt_fft(a_x, a_y, w);
  }
}

__global__ void FftWithTwiddle_Radix4(hipFloatComplex *a, hipFloatComplex *c, hipFloatComplex *t, const int N) {
  // shared memory
  __shared__ hipFloatComplex x[FFT1_SIZE];
  __shared__ hipFloatComplex constant[64];

  // global memory -> shared memory without shared memory bank conflict
  //x[threadIdx.x]                         = a[blockIdx.x * FFT1_SIZE + threadIdx.x];
  //x[threadIdx.x + BLOCK_DIM_1]           = a[blockIdx.x * FFT1_SIZE + BLOCK_DIM_1 + threadIdx.x];
  //x[threadIdx.x + 2 * BLOCK_DIM_1]       = a[blockIdx.x * FFT1_SIZE + 2 * BLOCK_DIM_1 + threadIdx.x];
  //x[threadIdx.x + 3 * BLOCK_DIM_1]       = a[blockIdx.x * FFT1_SIZE + 3 * BLOCK_DIM_1 + threadIdx.x];
  x[threadIdx.x]                           = a[blockIdx.x + FFT2_SIZE * (threadIdx.x)];
  x[threadIdx.x + BLOCK_DIM_1]             = a[blockIdx.x + FFT2_SIZE * (BLOCK_DIM_1 + threadIdx.x)];
  x[threadIdx.x + 2 * BLOCK_DIM_1]         = a[blockIdx.x + FFT2_SIZE * (2 * BLOCK_DIM_1 + threadIdx.x)];
  x[threadIdx.x + 3 * BLOCK_DIM_1]         = a[blockIdx.x + FFT2_SIZE * (3 * BLOCK_DIM_1 + threadIdx.x)];
  constant[threadIdx.x]                    = c[threadIdx.x];
  constant[threadIdx.x + blockDim.x]       = c[threadIdx.x + blockDim.x];
  __syncthreads();

  // FFT + Twiddle
  fft_radix4_odd(FFT1_SIZE, x, constant, t);

  // shared memory -> global memory without shared memory bank conflict
  //a[blockIdx.x * FFT1_SIZE + threadIdx.x]                   = x[threadIdx.x];
  //a[blockIdx.x * FFT1_SIZE + BLOCK_DIM_1 + threadIdx.x]     = x[threadIdx.x + BLOCK_DIM_1];
  //a[blockIdx.x * FFT1_SIZE + 2 * BLOCK_DIM_1 + threadIdx.x] = x[threadIdx.x + 2 * BLOCK_DIM_1];
  //a[blockIdx.x * FFT1_SIZE + 3 * BLOCK_DIM_1 + threadIdx.x] = x[threadIdx.x + 3 * BLOCK_DIM_1];
  a[blockIdx.x + FFT2_SIZE * (threadIdx.x)]                   = x[threadIdx.x];
  a[blockIdx.x + FFT2_SIZE * (BLOCK_DIM_1 + threadIdx.x)]     = x[threadIdx.x + BLOCK_DIM_1];
  a[blockIdx.x + FFT2_SIZE * (2 * BLOCK_DIM_1 + threadIdx.x)] = x[threadIdx.x + 2 * BLOCK_DIM_1];
  a[blockIdx.x + FFT2_SIZE * (3 * BLOCK_DIM_1 + threadIdx.x)] = x[threadIdx.x + 3 * BLOCK_DIM_1];
}

__global__ void FftWithoutTwiddle_Radix4(hipFloatComplex *a, hipFloatComplex *c) {
  // shared memory
  __shared__ hipFloatComplex x[FFT2_SIZE];
  __shared__ hipFloatComplex constant[64];

  // global memory -> shared memory without shared memory bank conflict
  x[threadIdx.x]                   = a[blockIdx.x * FFT2_SIZE + threadIdx.x];
  x[threadIdx.x + BLOCK_DIM_2]     = a[blockIdx.x * FFT2_SIZE + BLOCK_DIM_2 + threadIdx.x];
  x[threadIdx.x + 2 * BLOCK_DIM_2] = a[blockIdx.x * FFT2_SIZE + 2 * BLOCK_DIM_2 + threadIdx.x];
  x[threadIdx.x + 3 * BLOCK_DIM_2] = a[blockIdx.x * FFT2_SIZE + 3 * BLOCK_DIM_2 + threadIdx.x];
  constant[threadIdx.x]            = c[threadIdx.x];
  __syncthreads();

  // FFT
  fft_radix4_even(FFT2_SIZE, x, constant);

  // shared memory -> global memory without shared memory bank conflict
  a[blockIdx.x * FFT2_SIZE + threadIdx.x]                   = x[threadIdx.x];
  a[blockIdx.x * FFT2_SIZE + BLOCK_DIM_2 + threadIdx.x]     = x[threadIdx.x + BLOCK_DIM_2];
  a[blockIdx.x * FFT2_SIZE + 2 * BLOCK_DIM_2 + threadIdx.x] = x[threadIdx.x + 2 * BLOCK_DIM_2];
  a[blockIdx.x * FFT2_SIZE + 3 * BLOCK_DIM_2 + threadIdx.x] = x[threadIdx.x + 3 * BLOCK_DIM_2];
}

__global__ void FftStudent(hipFloatComplex *a, const int m, const int N) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (N / 2) * 1;
       i += blockDim.x * gridDim.x) {
    // index in N/2 range
    int N_idx = i % (N / 2);
    // i'th block
    int m_idx = N_idx / m;
    // base address
    hipFloatComplex *a_np = a;
    int t_idx = N_idx % m;
    hipFloatComplex *a_x = a_np + 2 * m_idx * m + t_idx;
    hipFloatComplex *a_y = a_x + m;
    hipFloatComplex w = twiddle(-M_PI * (double)t_idx / (double)m);
    butt_fft(a_x, a_y, w);
  }
}

__device__ void butt_ifft(hipFloatComplex *a, hipFloatComplex *b,
                          hipFloatComplex w) {
  hipFloatComplex T = hipCsubf(*a, *b);
  *a = hipCaddf(*a, *b);
  (*a).x /= 2.0;
  (*a).y /= 2.0;
  *b = hipCmulf(T, w);
  (*b).x /= 2.0;
  (*b).y /= 2.0;
}

__global__ void Ifft(hipFloatComplex *a, const int m, const int N) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (N / 2);
       i += blockDim.x * gridDim.x) {
    // index in N/2 range
    int N_idx = i % (N / 2);
    // i'th block
    int m_idx = N_idx / m;
    // base address
    hipFloatComplex *a_np = a;
    int t_idx = N_idx % m;
    hipFloatComplex *a_x = a_np + 2 * m_idx * m + t_idx;
    hipFloatComplex *a_y = a_x + m;
    hipFloatComplex w = twiddle(M_PI * (double)t_idx / (double)m);
    butt_ifft(a_x, a_y, w);
  }
}

__global__ void IfftStudent(hipFloatComplex *a, const int m, const int N) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (N / 2);
       i += blockDim.x * gridDim.x) {
    // index in N/2 range
    int N_idx = i % (N / 2);
    // i'th block
    int m_idx = N_idx / m;
    // base address
    hipFloatComplex *a_np = a;
    int t_idx = N_idx % m;
    hipFloatComplex *a_x = a_np + 2 * m_idx * m + t_idx;
    hipFloatComplex *a_y = a_x + m;
    hipFloatComplex w = twiddle(M_PI * (double)t_idx / (double)m);
    butt_ifft(a_x, a_y, w);
  }
}

__global__ void bitReverse(std::complex<float> *a, int N) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (N);
       i += blockDim.x * gridDim.x) {
    int logN = __log2f(N);
    int N_idx = i % N;
    std::complex<float> *a_x = a;
    int revN = __brev(N_idx) >> (32 - logN);
    if (revN > N_idx) {
      std::complex<float> temp = a_x[N_idx];
      a_x[N_idx] = a_x[revN];
      a_x[revN] = temp;
    }
  }
}

__device__ hipFloatComplex Cmul(hipFloatComplex a, hipFloatComplex b) {
  float temp = double(a.x) * b.x - double(a.y) * b.y;
  float temp2 = double(a.x) * b.y + double(a.y) * b.x;
  hipFloatComplex res;
  res.x = temp;
  res.y = temp2;
  return res;
}

__global__ void Hadamard(hipFloatComplex *a, hipFloatComplex *b, int N) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (N);
       i += blockDim.x * gridDim.x) {
    int N_idx = i % N;
    hipFloatComplex *a_x = a;
    hipFloatComplex *b_x = b;
    a_x[N_idx] = Cmul(a_x[N_idx], b_x[N_idx]);
  }
}

void FftHelper::ExecFft(std::complex<float> *a, int N) {
  dim3 blockDim(refft::FFTblocksize);
  dim3 gridDim(N/2/refft::FFTblocksize);
  bitReverse<<<gridDim, blockDim>>>(a,N);
  for (int i = 1; i < N; i *= 2) {
    Fft<<<gridDim, blockDim>>>((hipFloatComplex *)a, i, N);
    CudaCheckError();
  }
  CudaCheckError();
}

void FftHelper::ExecStudentFft(std::complex<float> *a, std::complex<float> *c, std::complex<float> *t, int N){
  dim3 blockDim1(BLOCK_DIM_1);
  dim3 gridDim1(GRID_DIM_1);
  dim3 blockDim2(BLOCK_DIM_2);
  dim3 gridDim2(GRID_DIM_2);
  dim3 gridDim3(T_nx/T_SMEM_SIZE, T_ny/T_SMEM_SIZE, 1);
  dim3 blockDim3(T_SMEM_SIZE, T_BLOCK_ROW, 1);
  
  Cal<<<64, 1>>>((hipFloatComplex *)c, 128);
  Cal<<<TWIDDLE_GRID_DIM, TWIDDLE_BLOCK_DIM>>>((hipFloatComplex *)t, N/2);
  FftWithTwiddle_Radix4<<<gridDim1, blockDim1>>>((hipFloatComplex *)a, (hipFloatComplex *)c, (hipFloatComplex *)t, N);
  FftWithoutTwiddle_Radix4<<<gridDim2, blockDim2>>>((hipFloatComplex *)a, (hipFloatComplex *)c);
  CudaCheckError();
  Transpose<<<gridDim3, blockDim3>>>((hipFloatComplex *)a);
  CudaCheckError();
}

void FftHelper::ExecIfft(std::complex<float> *a, int N) {
  dim3 blockDim(refft::iFFTblocksize);
  dim3 gridDim(N/2/refft::iFFTblocksize);
  for (int i = N / 2; i > 0; i >>= 1) {
    Ifft<<<gridDim, blockDim>>>((hipFloatComplex *)a, i, N);
  }
  bitReverse<<<gridDim, blockDim>>>(a, N);
  CudaCheckError();
}

void FftHelper::ExecStudentIfft(std::complex<float> *a, int N) {
  dim3 blockDim(refft::iFFTblocksize);
  dim3 gridDim(N/2/refft::iFFTblocksize);
  for (int i = N / 2; i > 0; i >>= 1) {
    IfftStudent<<<gridDim, blockDim>>>((hipFloatComplex *)a, i, N);
  }
  bitReverse<<<gridDim, blockDim>>>(a, N);
  CudaCheckError();
}

void FftHelper::Mult(std::complex<float> *a, std::complex<float> *b, int N) {
  dim3 blockDim(refft::iFFTblocksize);
  dim3 gridDim(N/refft::iFFTblocksize);
  Hadamard<<<gridDim, blockDim>>>((hipFloatComplex*)a,(hipFloatComplex*)b, N);  
  CudaCheckError();
}
}  // namespace refft
